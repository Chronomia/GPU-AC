#include "hip/hip_runtime.h"
# include <iostream>
# include <stdio.h>
# include <string>
# include <vector>
# include <fstream>
# include <cmath>
# include <algorithm>
# include <unordered_map>
# include <set>
# include <thrust/device_vector.h>

# include "../../headers/insert_sequence/tools.h"
# include "../../headers/insert_sequence/record.h"

# define NEW_STACK_SIZE 5000
# define BUILD_BLOCK_SIZE 512
# define QUERY_BLOCK_SIZE 512
// # define DEBUG

using namespace::std;

unsigned long *bitmap;

// cpu data structures
int rules_content_size;
vector<int> word_lengths;
vector<int> rules_content;

// query
int* query_file_indexes;
int* query_insert_offsets;
string insert_strings;
vector<struct insert_query>* insert_querys;

// insert records
__device__ char* insert_strings_device;
__device__ int* string_start_indexes_device;
__device__ struct record* records_device;
__device__ unsigned long long* element_bitmap_device;
__device__ int curr_record_num_device;
__device__ int* relation_map_device;
__device__ int* insert_file_split_indexes_device;

// gpu data structures
__constant__ int word_num_device, rule_num_device, file_split_num_device;
__constant__ int hashtable_size_device;
__constant__ int records_size_device;

__device__ int* rules_content_device;
__device__ int* word_lengths_device;
__device__ int* rule_lengths_device;

__device__ int* rule_split_indexes_device;
__device__ int* file_split_indexes_device;

__device__ unsigned int* root_rule_start_offsets_device;


// functions
extern __global__ void insert(int *file_indexes, int offsets, int query_num);
__global__ void get_offsets(int root_rule_size, int* root_rule_offsets_device);

int main(int argc, char** argv){
	int file_split_num;
	
	// --- IO --- //

	clock_t time1 = clock();
	string input_file_path = argv[1];

	// process fileYyNO.txt

	string fileYyNo_path = input_file_path + "fileYyNO.txt";
	ifstream fin_filesplit(fileYyNo_path);
	check_fin(fin_filesplit, fileYyNo_path);
	
	fin_filesplit >> file_split_num;
	hipMemcpyToSymbol(HIP_SYMBOL(file_split_num_device), &file_split_num, sizeof(int));

	int file_split_word[file_split_num];
	int temp = 0;
	for(int i = 0; i < file_split_num; i ++){	
		fin_filesplit >> temp >> file_split_word[i];
	}

	fin_filesplit.close();


	// process rowCol.dic

	string rowCol_path = input_file_path + "rowCol.dic";
	ifstream fin_rules(rowCol_path);
	check_fin(fin_rules, rowCol_path);

	int word_num, rule_num = 0;
	fin_rules >> word_num >> rule_num;
	hipMemcpyToSymbol(HIP_SYMBOL(word_num_device), &word_num, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(rule_num_device), &rule_num, sizeof(int));
	cout << "word number : " << word_num << ", rule number : " << rule_num << ", file number : "<< file_split_num << endl;

	int rule_size, element;
	int* rule_split_indexes = (int* )malloc(sizeof(int) * (rule_num + 1));
	rule_split_indexes[0] = 0;

	for(int i = 1; i <= rule_num; i ++){
		fin_rules >> rule_size;
		rule_split_indexes[i] = rule_size + rule_split_indexes[i - 1];

		for(int j = 0; j < rule_size; j ++){
			fin_rules >> element;
			rules_content.push_back(element);
		}
	}

	rules_content_size = rules_content.size();
	fin_rules.close();


	// process dictionary.dic

	string dictionary_path = input_file_path + "dictionary.dic";
	ifstream fin_worddic(dictionary_path);
	check_fin(fin_worddic, dictionary_path);

	int word_index;
	string word;
	string *word_collection = new string[word_num];

	while(!fin_worddic.eof()){
		fin_worddic >> word_index;
		fin_worddic.get();
		// fin_worddic >> word; // not working 
		word = get_word(fin_worddic); // super nb
	
		word_collection[word_index] = word;
		word_lengths.push_back(word.length());
	}

	fin_worddic.close();

	// get file split index

	int split_curr = 0;
	int* file_split_indexes = (int* )malloc(sizeof(int) * (file_split_num + 1));
	file_split_indexes[0] = 0;

	for(int i = rule_split_indexes[0]; i < rule_split_indexes[1]; i ++){
		if(rules_content[i] == file_split_word[split_curr] && split_curr < file_split_num){
			file_split_indexes[split_curr + 1] = i;
			split_curr ++;
		}
	}
	int root_rule_size = file_split_indexes[file_split_num];

	clock_t time2;
	time2 = clock();
	cout << endl;
	cout << "IO time : " << (double)(time2 - time1) / CLOCKS_PER_SEC << "s" << endl;
	cout << "===============" << endl;

	// --- insert query --- //

	int query_size;
	
	string query_path = "../../query/insert_1_query.txt";
	ifstream fin_query(query_path);
	check_fin(fin_query, query_path);

	fin_query >> query_size;
	// cout << "total query size : " << query_size << endl;

	int query_malloc_size = query_size * sizeof(int);

	query_file_indexes = (int *)malloc(query_malloc_size);
	query_insert_offsets = (int* )malloc(query_malloc_size);
	int* insert_string_lengths = (int* )malloc(query_malloc_size);

	insert_querys = new vector<struct insert_query> [file_split_num]; // in files

	int file_index, offset;
	string temp_string;
	int insert_string_total_length;

	insert_strings = "";

	for(int i = 0; i < query_size; i ++){
		fin_query >> file_index >> offset >> temp_string;

		// update struct
		struct insert_query temp_insert_query;
		temp_insert_query.file_index = file_index;
		temp_insert_query.insert_offset = offset;
		temp_insert_query.string_index = i;

		insert_querys[file_index].push_back(temp_insert_query);


		query_file_indexes[i] = file_index;
		query_insert_offsets[i] = offset;
		insert_strings += temp_string;
		
		int string_length = temp_string.length();
		insert_string_lengths[i] = string_length;
		insert_string_total_length += string_length;

		// cout << file_index << " " << temp_string << endl;
	}
	fin_query.close();
	//printf("haha\n");
	// --- process insert queries --- //
	int* insert_file_split_indexes = (int* )malloc(sizeof(int) * (file_split_num + 1));
	insert_file_split_indexes[0] = 0;

	// put queries into sequence by file index
	int* new_query_file_indexes = (int* )malloc(query_malloc_size);
	int* new_query_insert_offsets = (int* )malloc(query_malloc_size);
	int* new_string_indexes = (int* )malloc(query_malloc_size);

	int insert_index = 0;
	for(int i = 0; i < file_split_num; i ++){
		insert_file_split_indexes[i + 1] = insert_file_split_indexes[i] + insert_querys[i].size(); 
		
		for(auto it : insert_querys[i]){
			new_query_file_indexes[insert_index] = it.file_index;
			new_query_insert_offsets[insert_index] = it.insert_offset;
			new_string_indexes[insert_index] = it.string_index;
			insert_index += 1;
		}
	}


	// --- malloc for insert --- //

	// rules content in device
	int* temp_rules_malloc;
	hipMalloc(&temp_rules_malloc, sizeof(int) * rules_content_size);
	hipMemcpy(temp_rules_malloc, &rules_content[0], sizeof(int) * rules_content_size, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(rules_content_device), &temp_rules_malloc, sizeof(temp_rules_malloc));

	// file and rule split indexes (start from 0)
	int* temp_rule_split_indexes_device;
	int* temp_file_split_indexes_device;

	hipMalloc(&temp_rule_split_indexes_device, sizeof(int) * (rule_num + 1));
	hipMalloc(&temp_file_split_indexes_device, sizeof(int) * (file_split_num + 1));

	hipMemcpy(temp_rule_split_indexes_device, rule_split_indexes, sizeof(int) * (rule_num + 1), hipMemcpyHostToDevice);
	hipMemcpy(temp_file_split_indexes_device, file_split_indexes, sizeof(int) * (file_split_num + 1), hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(rule_split_indexes_device), &temp_rule_split_indexes_device, sizeof(temp_rule_split_indexes_device));
	hipMemcpyToSymbol(HIP_SYMBOL(file_split_indexes_device), &temp_file_split_indexes_device, sizeof(temp_file_split_indexes_device));

	// load file indexes to device
	int *query_file_indexes_device;
	hipMalloc(&query_file_indexes_device, query_malloc_size);
	hipMemcpy(query_file_indexes_device, new_query_file_indexes, query_malloc_size, hipMemcpyHostToDevice);
	// load insert offsets to device
	int *query_insert_offsets_device;
	hipMalloc(&query_insert_offsets_device, query_malloc_size);
	hipMemcpy(query_insert_offsets_device, new_query_insert_offsets, query_malloc_size, hipMemcpyHostToDevice);
	// load string indexes to device
	int *insert_string_indexes_device;
	hipMalloc(&insert_string_indexes_device, query_malloc_size);
	hipMemcpy(insert_string_indexes_device, new_string_indexes, query_malloc_size, hipMemcpyHostToDevice);
	// insert file split indexes
	int *temp_insert_file_split_indexes;
	hipMalloc(&temp_insert_file_split_indexes, sizeof(int) * (file_split_num + 1));
	hipMemcpy(temp_insert_file_split_indexes, insert_file_split_indexes, sizeof(int) * (file_split_num + 1), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(insert_file_split_indexes_device), &temp_insert_file_split_indexes, sizeof(temp_insert_file_split_indexes));

	char* temp_insert_strings_device;
	hipMalloc(&temp_insert_strings_device, sizeof(char) * insert_string_total_length);
	hipMemcpy(temp_insert_strings_device, &insert_strings[0], sizeof(char) * insert_string_total_length, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(insert_strings_device), &temp_insert_strings_device, sizeof(temp_insert_strings_device));

	int* string_split_indexes = (int* )malloc(sizeof(int) * (query_size + 1));
	aggregate_index(insert_string_lengths, string_split_indexes, query_size);
	
	int* temp_string_start_indexes;
	hipMalloc(&temp_string_start_indexes, sizeof(int) * (query_size + 1));
	hipMemset(temp_string_start_indexes, 0x00, sizeof(int) * (query_size + 1));
	hipMemcpyToSymbol(HIP_SYMBOL(string_start_indexes_device), &temp_string_start_indexes, sizeof(temp_string_start_indexes));

	// records on device
	struct record* temp_records = create_record_set(query_size);
	hipMemcpyToSymbol(HIP_SYMBOL(records_device), &temp_records, sizeof(temp_records));

	// bitmap on device
	// cout << "size of unsigned long long : " << sizeof(unsigned long long) << endl;
	unsigned long long* temp_element_bitmap;
	int bitmap_size = rule_split_indexes[rule_num];
	hipMalloc(&temp_element_bitmap, sizeof(unsigned long long) * ((bitmap_size >> 6) + 1));
	hipMemset(temp_element_bitmap, 0x00, sizeof(unsigned long long) * ((bitmap_size >> 6) + 1));
	hipMemcpyToSymbol(HIP_SYMBOL(element_bitmap_device), &temp_element_bitmap, sizeof(temp_element_bitmap));

	int* temp_word_lengths_device;
	hipMalloc(&temp_word_lengths_device, sizeof(int) * word_num);
	hipMemcpy(temp_word_lengths_device, &word_lengths[0], sizeof(int) * word_num, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(word_lengths_device), &temp_word_lengths_device, sizeof(temp_word_lengths_device));

	int* temp_rule_lengths_device;
	hipMalloc(&temp_rule_lengths_device, sizeof(int) * rule_num);
	hipMemset(temp_rule_lengths_device, 0x00, sizeof(int) * rule_num);
	hipMemcpyToSymbol(HIP_SYMBOL(rule_lengths_device), &temp_rule_lengths_device, sizeof(temp_rule_lengths_device));

	// root rule offsets:
	int* temp_root_rule_offsets;
	hipMalloc(&temp_root_rule_offsets, sizeof(int) * root_rule_size);
	// hipMemcpyToSymbol(HIP_SYMBOL(root_rule_offsets_device), &temp_root_rule_offsets, sizeof(temp_root_rule_offsets)); 

	// --- run dfs --- //

	hipError_t stat;

	// check_gpu_mem();
	stat = hipDeviceSetLimit(hipLimitStackSize, NEW_STACK_SIZE);
	cout << "if stack successfully allocated : " << (stat == 0) << endl;

	int build_block_size = BUILD_BLOCK_SIZE;
	int build_grid_size = (root_rule_size + build_block_size - 1) / build_block_size;


	get_offsets<<<build_grid_size, build_block_size>>>(root_rule_size, temp_root_rule_offsets);
	hipDeviceSynchronize();

	// # ifdef DEBUG
	// printf("after get offset : %s\n",hipGetErrorString(hipGetLastError()));
	// # endif

	clock_t time3 = clock();
	cout << endl;
	cout << "GPU building time : " << (double)(time3 - time2) / CLOCKS_PER_SEC << "s" << endl;
	cout << "===============" << endl;

	// copy back offsets and aggregrate
	int* root_rule_offsets_host = (int* )malloc(sizeof(int) * root_rule_size);
	stat = hipMemcpy(root_rule_offsets_host, temp_root_rule_offsets, sizeof(int) * root_rule_size, hipMemcpyDeviceToHost);
	
	int* root_rule_start_offsets = (int* )malloc(sizeof(int) * (root_rule_size + 1));
	// aggregate_index(root_rule_offsets_host, root_rule_start_offsets, root_rule_size);
	aggregate_index_for_fileoffsets(root_rule_offsets_host, root_rule_start_offsets, root_rule_size, file_split_indexes, file_split_num);

	// set device int* 
	unsigned int* temp_root_rule_start_offsets;
	hipMalloc(&temp_root_rule_start_offsets, sizeof(unsigned int) * (root_rule_size + 1));
	hipMemcpy(temp_root_rule_start_offsets, root_rule_start_offsets, sizeof(unsigned int) * (root_rule_size + 1), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(root_rule_start_offsets_device), &temp_root_rule_start_offsets, sizeof(temp_root_rule_start_offsets));

	// set curr record number
	int temp_curr_record_num = 0;
	hipMemcpyToSymbol(HIP_SYMBOL(curr_record_num_device), &temp_curr_record_num, sizeof(int));

	// relation_map
	int* relation_map;
	hipMalloc(&relation_map, sizeof(int) * bitmap_size);
	hipMemset(relation_map, 0x00, sizeof(int) * bitmap_size);
	hipMemcpyToSymbol(HIP_SYMBOL(relation_map_device), &relation_map, sizeof(relation_map));

	hipFree(temp_root_rule_offsets);
	free(root_rule_offsets_host);
	free(root_rule_start_offsets);

	// --- run insert --- //

	int insert_block_size = QUERY_BLOCK_SIZE;
	int insert_grid_size  = (file_split_num + insert_block_size - 1) / insert_block_size; // split by file

	# ifdef DEBUG
	printf("before insert : %s\n",hipGetErrorString(hipGetLastError()));
	# endif

	clock_t time4 = clock();
	insert<<<insert_grid_size, insert_block_size>>>(query_file_indexes_device, query_insert_offsets_device, insert_string_indexes_device);
	hipDeviceSynchronize();

	# ifdef DEBUG
	printf("after insert : %s\n",hipGetErrorString(hipGetLastError()));
	# endif

	clock_t time5 = clock();
	cout << endl;
	cout << "INSERT time(s): " << (double)(time5 - time4) / CLOCKS_PER_SEC << endl;
	cout << "AVGLatency(s): " << (double)(time5 - time4) / CLOCKS_PER_SEC / query_size << endl;
    cout << "AVGLatency(us): " << (double)(time5 - time4) / CLOCKS_PER_SEC / query_size * 1000000
         << endl;
    cout << "Throughput(op/s): " << query_size * CLOCKS_PER_SEC / (double)(time5 - time4)
         << endl;
	cout << "===============" << endl;
	


	return 0;
}


__device__ int dfs_for_offset(int element){
	if(element < word_num_device){
		return word_lengths_device[element];
	}
	else{
		int rule_index = element - word_num_device;
		int rule_start_index = rule_split_indexes_device[rule_index];
		int rule_end_index = rule_split_indexes_device[rule_index + 1];

		int temp_curr_offset = 0;
		for(int i = rule_start_index; i < rule_end_index; i ++){
			int temp_element = rules_content_device[i];
			int subtree_offset = dfs_for_offset(temp_element);

			temp_curr_offset += subtree_offset;
		}
		return temp_curr_offset;
	}

	return 0;
}

__global__ void get_offsets(int root_rule_size, int* root_rule_offsets_device){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid >= root_rule_size){
		return;
	}

	int element = rules_content_device[tid];

	int element_offset = dfs_for_offset(element);
	root_rule_offsets_device[tid] = element_offset;
}
