#include "hip/hip_runtime.h"
# include <iostream>
# include <stdio.h>
# include <string>
# include <vector>

# include "../../headers/insert_batch/record.h"

# define BLOCK_SIZE 512

extern __constant__ int records_size_device;
extern __constant__ int word_num_device, file_split_num_device;

extern __device__ struct record* records_device;
extern __device__ int* rules_content_device;
extern __device__ int* word_lengths_device;

extern __device__ int* string_start_indexes_device;
extern __device__ char* insert_strings_device;

extern __device__ unsigned int* root_rule_start_offsets_device;
extern __device__ unsigned long long* element_bitmap_device;
extern __device__ int* file_split_indexes_device;
extern __device__ int* rule_split_indexes_device;
extern __device__ int curr_record_num_device;
extern __device__ int* relation_map_device;

struct record* create_record_set(int size){
	int malloc_size = sizeof(struct record) * size;
	// cout << "malloc size is : " << malloc_size << endl;
	struct record* temp_records_device;
	hipError_t stat;

	stat = hipMalloc(&temp_records_device, malloc_size);
	hipMemset(temp_records_device, 0x00, malloc_size);

	if(stat){
		cout << endl;
		cout << "cudamalloc records failed with stat : " << stat << endl;
		return NULL;
	}
	// stat = hipMemcpyToSymbol(HIP_SYMBOL(records_device), &temp_records_device, sizeof(temp_records_device));
	hipMemcpyToSymbol(HIP_SYMBOL(records_size_device), &size, sizeof(int));
	
	return temp_records_device;
}

__global__ void insert(int* file_indexes, int* insert_offsets, struct insert_update_record* insert_update_records_device, int query_num){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid >= query_num){
		return;
	}
	int file_index = file_indexes[tid];
	int insert_offset = insert_offsets[tid];

	int file_start_index = file_split_indexes_device[file_index];
	int file_end_index   = file_split_indexes_device[file_index + 1];
	// printf("%d %d\n", file_index, file_split_indexes_device[1]);

	if((insert_offsets < 0) || (root_rule_start_offsets_device[file_end_index - 1] <= insert_offset)){
		// printf("%d %d\n", file_end_index - 1, root_rule_start_offsets_device[file_end_index - 1]);
		printf("insert invalid. \n");
	}

	int search_start = file_start_index;
	int search_end   = file_end_index;
	int search_mid   = (search_start + search_end) / 2;

	while(root_rule_start_offsets_device[search_mid] > insert_offset || 
		  root_rule_start_offsets_device[search_mid + 1] <= insert_offset){
		if(search_start == search_mid){
			break;
		}

		// int last_search_start = search_start;
		// int last_search_end   = search_end;
		int last_search_mid   = search_mid;

		if(insert_offset < root_rule_start_offsets_device[search_mid]){
			search_end = last_search_mid - 1;
		}	
		else{
			search_start = last_search_mid;
		}
		search_mid = (search_start + search_end) / 2;
	}

	int root_rule_index =  search_mid; // the index in root rule to insert
	int element_index = rules_content_device[search_mid];

	if(element_index < word_num_device){
		// if element is word
		struct record temp_record;
		if(element_bitmap_device[root_rule_index >> 6] & (1ul << (root_rule_index & 0x3f))){
			temp_record.no = relation_map_device[root_rule_index];
		}
		else{
			element_bitmap_device[root_rule_index >> 6] = 
			element_bitmap_device[root_rule_index >> 6] & (1ul << (root_rule_index & 0x3f));
			temp_record.no = -1;
		}

		temp_record.file_index = file_index;
		temp_record.file_offset = insert_offset;
		temp_record.rule_index = 0;
		temp_record.rule_start_offset = 0;
		temp_record.rule_location = root_rule_index;
		temp_record.replace_word = element_index; // original word
		temp_record.content = &insert_strings_device[string_start_indexes_device[tid]];
		int insert_string_length = string_start_indexes_device[tid] - string_start_indexes_device[tid + 1];
		temp_record.content_length = insert_string_length;

		int record_index = atomicAdd(&curr_record_num_device, 1);

		records_device[record_index] = temp_record;
		relation_map_device[root_rule_index] = record_index;	

		struct insert_update_record temp_update_record;
		temp_update_record.file_index = file_index;
		temp_update_record.insert_offset = insert_offset;
		temp_update_record.root_insert_index = root_rule_index;
		temp_update_record.insert_length = insert_string_length;
		insert_update_records_device[record_index] = temp_update_record;

		// no update 
	}
	else{
		// if element is rule
		int curr_offset = root_rule_start_offsets_device[search_mid];
		insert_into_rule(element_index - word_num_device, search_mid, curr_offset, file_index, insert_offset, insert_update_records_device, tid);
	}

	return;
}


__device__ int insert_into_rule(int rule_index, int insert_index, int& curr_offset, int file_index, int insert_offset, struct insert_update_record* insert_update_records_device, int tid){ // search mid is insert index(root rule index)
	// int file_end_index = file_split_indexes_device[file_index + 1];
	int rule_start_offset = curr_offset; // curr offset is the start offset of rule to insert

	int rule_start_index = rule_split_indexes_device[rule_index];
	int rule_end_index = rule_split_indexes_device[rule_index + 1];

	for(int i = rule_start_index; i < rule_end_index; i ++){
		int element_index = rules_content_device[i];
		if(element_index < word_num_device && !(element_bitmap_device[i >> 6] & (1ul << (i & 0x3f)))){
			// if is word and bit map not set
			int new_offset = curr_offset + word_lengths_device[element_index];
			if(insert_offset > new_offset){
				// keep searching
				curr_offset = new_offset;
				continue;
			}
			else{ // end searching
				struct record temp_record;
				element_bitmap_device[(i >> 6)] = element_bitmap_device[(i >> 6)] | (1ul << (i & 0x3f));
				temp_record.no = -1;

				// get record_index
				int insert_record_index = atomicAdd(&curr_record_num_device, 1);
				relation_map_device[i] = insert_record_index;

				temp_record.file_index = file_index;
				temp_record.file_offset = insert_offset;
				temp_record.rule_start_offset = rule_start_offset;
				temp_record.rule_index = rule_index;
				temp_record.rule_location = i;
				temp_record.replace_word = element_index;
				temp_record.content = &insert_strings_device[string_start_indexes_device[tid]];
				int insert_string_length = string_start_indexes_device[tid + 1] - string_start_indexes_device[tid];
				temp_record.content_length = insert_string_length;

				records_device[insert_record_index] = temp_record;

				struct insert_update_record temp_update_record;
				temp_update_record.file_index = file_index;
				temp_update_record.insert_offset = insert_offset;
				temp_update_record.root_insert_index = insert_index;
				temp_update_record.insert_length = insert_string_length;
				insert_update_records_device[insert_record_index] = temp_update_record;

				// break;
				return 1; // inserted, for end
			}
			
		}
		else if(element_index < word_num_device && element_bitmap_device[i >> 6] && (1ul << (i & 0x3f))){ // if is word but bitmap set
			int record_index = relation_map_device[i]; // the last record index of this rule
			int content_size = 0;

			struct record temp_record = records_device[record_index];
			if(temp_record.file_index == file_index &&
				temp_record.rule_start_offset == rule_start_offset){ // if in the same location
				content_size += temp_record.content_length;
			}	

			while(temp_record.no >= 0){ // view back every record in the same location
				record_index = temp_record.no;
				// set new temp record
				temp_record = records_device[record_index];
				if(temp_record.file_index == file_index && 
					temp_record.rule_start_offset == rule_start_offset){
					content_size += temp_record.content_length;
				}

			}

			content_size += word_lengths_device[records_device[record_index].replace_word];
		
			int new_offset = curr_offset + content_size;
			if(insert_offset > new_offset){ // keep searching
				curr_offset = new_offset;
				continue;
			}
			else{ // end searching
				struct record insert_record;
				int insert_record_index = atomicAdd(&curr_record_num_device, 1);

				insert_record.no = relation_map_device[i];
				relation_map_device[i] = insert_record_index;

				insert_record.file_index = file_index;
				insert_record.file_offset = insert_offset;
				insert_record.rule_start_offset = rule_start_offset;
				insert_record.rule_index = rule_index;
				insert_record.rule_location = i;
				insert_record.replace_word = element_index;
				insert_record.content = &insert_strings_device[string_start_indexes_device[tid]];
				int insert_string_length = string_start_indexes_device[tid + 1] - string_start_indexes_device[tid];
				insert_record.content_length = insert_string_length;

				records_device[insert_record_index] = insert_record;

				struct insert_update_record temp_update_record;
				temp_update_record.file_index = file_index;
				temp_update_record.insert_offset = insert_offset;
				temp_update_record.root_insert_index = insert_index;
				temp_update_record.insert_length = insert_string_length;
				insert_update_records_device[record_index] = temp_update_record;

				// to update root offset and record offset
				// for(int j = insert_index + 1; j < file_end_index; j ++){
				// 	// root_rule_start_offsets_device[j] += insert_string_length;
				// 	atomicAdd(&root_rule_start_offsets_device[j], insert_string_length);
				// }
				// for(int j = 0; j < insert_record_index; j ++){
				// 	struct record search_record = records_device[j];
				// 	if(file_index == search_record.file_index){
				// 		// if record file index hits, update file offset
				// 		if(search_record.file_offset > insert_offset){
				// 			// records_device[j].file_offset += insert_string_length;
				// 			atomicAdd(&records_device[j].file_offset, insert_string_length);
				// 		}
				// 		if(search_record.rule_start_offset > insert_offset){
				// 			// records.device[j].rule_start_offset += insert_string_length;
				// 			atomicAdd(&records_device[j].rule_start_offset, insert_string_length);
				// 		}
				// 	}
				// }

				return 1; // record inserted
			}
			
		}
		else if(element_index >= word_num_device){
			// if is still rule
			int if_inserted = insert_into_rule(element_index - word_num_device, insert_index, curr_offset, file_index, insert_offset, insert_update_records_device, tid);
			if(if_inserted){
				return 1;
			}
		}
	}
	
	return 0; // if not inserted
}

__device__ int get_file_index(int root_rule_index){
	int file_index = 0;
	
	for(int i = 0; i < file_split_num_device; i ++){
		if(root_rule_index < file_split_indexes_device[i + 1]){
			break;
		}
		file_index += 1;
	}

	return file_index;
}

__global__ void update_root_start_offsets(int range, int record_index, int tid_file_index, struct insert_update_record* insert_update_records_device){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid >= range){
		return;
	}

	struct insert_update_record temp_update_record = insert_update_records_device[tid];
	int file_index = temp_update_record.file_index;
	int root_insert_index = temp_update_record.root_insert_index;
	int insert_length = temp_update_record.insert_length;

	if(root_insert_index < record_index && file_index == tid_file_index){
		// root_rule_start_offsets_device[tid] += offset_update;
		atomicAdd(&root_rule_start_offsets_device[record_index], insert_length);
	}

}

__global__ void insert_update_offsets(struct insert_update_record* insert_update_records_device, int root_size){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid >= root_size){
		return;
	}

	int tid_file_index = get_file_index(tid);
	int offset_update = 0;

	// int block_size = BLOCK_SIZE;
	// int grid_size  = (records_size_device + block_size - 1) / block_size;

	// update_root_start_offsets<<<grid_size, block_size>>>(records_size_device, tid, tid_file_index, insert_update_records_device);

	// to process the root_rule_start_offset[tid]
	for(int i = 0; i < records_size_device; i ++){
		struct insert_update_record temp_update_record = insert_update_records_device[i];
		int file_index = temp_update_record.file_index;
		int root_insert_index = temp_update_record.root_insert_index;
		int insert_length = temp_update_record.insert_length;

		if(root_insert_index < tid && file_index == tid_file_index){
			offset_update += insert_length;
		}
	}

	root_rule_start_offsets_device[tid] += offset_update;

	return;

}

__global__ void insert_update_records(struct insert_update_record* insert_update_records_device, int query_size){
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid >= query_size){
		return;
	}

	struct record this_record = records_device[tid];
	struct insert_update_record this_update_record = insert_update_records_device[tid];

	// to process the records[tid]

	for(int i = 0; i < records_size_device; i ++){
		struct insert_update_record temp_update_record = insert_update_records_device[i];

		if(temp_update_record.file_index == this_update_record.file_index){

			if(temp_update_record.insert_offset < this_update_record.insert_offset){
				this_record.file_offset += temp_update_record.insert_length;
			}
			if(temp_update_record.insert_offset < root_rule_start_offsets_device[this_update_record.root_insert_index]){
				this_record.rule_start_offset += temp_update_record.insert_length;
			}
			
		}
	}

	records_device[tid] = this_record;

	return;
}